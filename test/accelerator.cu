//==================================================================================================
/**
  SPY - C++ Informations Broker
  Copyright : SPY Project Contributors
  SPDX-License-Identifier: BSL-1.0
**/
//==================================================================================================
#include <spy.hpp>
#include <iostream>

int main()
{
  std::cout << "Check that specified accelerator is supported: " << std::endl;
  {
    #if defined(SYCL_LANGUAGE_VERSION) && defined (__INTEL_LLVM_COMPILER)
    static_assert( spy::supports::sycl );
    std::cout << "Currently compiling with " << spy::supports::sycl << " enabled\n";
    #else
    static_assert( !spy::supports::sycl );
    std::cout << "Currently compiling without SYCL enabled\n";
    #endif
  }
  {
    #if defined(__NVCC__) && defined (__HIPCC__)
    static_assert( spy::supports::cuda );
    std::cout << "Currently compiling with " << spy::supports::cuda << " enabled\n";
    #else
    static_assert( !spy::supports::cuda );
    std::cout << "Currently compiling without CUDA enabled\n";
    #endif
  }
  std::cout << "Done." << std::endl;
}
